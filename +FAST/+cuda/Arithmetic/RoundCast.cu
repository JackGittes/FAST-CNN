
#include <hip/hip_runtime.h>
/********************************************************
*	Author: Zhao Mingxin
*	Date:	2020/11/10
*	Description: CUDA Kernel for mul-shift-round operation.
*	The input x will be multiplied with mul, then right-shift
*	shift-1 bits. Rounding up or down is determined by the 
*	remaining last bit, which means we round down if the 
*	remaining bit is 0.
*
*	NOTE:	If you have any issues about this code, please
*	feedback.
*	Homepage:	https://jackgittes.github.io
*********************************************************/

__global__ void RoundCast(const long long *src, long long *dst,
                        const int mul, const int shift, const int bit_width,
                        const int height, const int width, const int chn)
{
    long long tmp;
    long long last_bit_mask = 1;
    long long bit_value;
    long long upbound = 2 << (bit_width-1) - 1, lowbound = -2 << (bit_width-1);

    // Calculate current element id.
    int chn_id = blockIdx.z;
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;
    int col_id = blockIdx.y * blockDim.y + threadIdx.y;
    int ele_id = chn_id * height * width + col_id * height + row_id;

    if(row_id<width && col_id<height){
        tmp = src[ele_id];

        tmp = (tmp * mul) >> (shift-1);
        bit_value = tmp & last_bit_mask;
        tmp = (tmp >> 1) + bit_value; 

        tmp = tmp > upbound ? upbound:tmp;
        tmp = tmp < lowbound ? lowbound:tmp;

        dst[ele_id] = tmp;
    }
}
