
#include <hip/hip_runtime.h>
/********************************************************
*	Author: Zhao Mingxin
*	Date:	2018/12/11
*	Description: CUDA Kernel for matmul. 
*
*	NOTE:	If you have any issues about this code, please
*	feedback.
*	Homepage:	https://jackgittes.github.io
*********************************************************/
__global__ void MatMulKernel(const long long *A,const long long *B,const int Aheight,const int Awidth,const int Bwidth, const long long up_bound,const long long low_bound,long long *C)
{
	long long Cvalue = 0;
	long long prod_tmp;
	int Bheight = Awidth;
	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int e = 0; e < Awidth; ++e){
		prod_tmp = A[Aheight * e + row]*B[col * Bheight + e];
		if(prod_tmp>up_bound)
			prod_tmp=up_bound;
		if(prod_tmp<low_bound)
			prod_tmp=low_bound;
		
		Cvalue+=prod_tmp;
		if(Cvalue>up_bound)
			Cvalue=up_bound;
		if(Cvalue<low_bound)
			Cvalue=low_bound;
	}	
	C[Aheight*col + row] = Cvalue;
}